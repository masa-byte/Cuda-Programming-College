#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <math.h>
#include <map>
#include <sys/time.h>

using namespace std;

#define TOTAL_DEGREES 180
#define BINS_PER_DEGREE 4
#define TOTAL_BINS (TOTAL_DEGREES * BINS_PER_DEGREE + 1) // + 1 for the last degree bin
#define PARAMETERS 2

const float ARCMIN_TO_RADIAN = M_PI / (180 * 60);

const int NUM_THREADS = 256;

__device__ float angleBetweenGalaxies(float rightAscension1, float declination1, float rightAscension2, float declination2)
{
    float expression = sin(declination1) * sin(declination2) + cos(declination1) * cos(declination2) * cos(rightAscension1 - rightAscension2);

    if (expression > 1)
        expression = 1.0;
    else if (expression < -1)
        expression = -1.0;

    return acos(expression);
}

__global__ void calculateHistogram(float *galaxy1, float *galaxy2, int n, unsigned int *histogramBins)
{
    // Each thread will cover n galaxies
    // Galaxy1 is fixed and galaxy2 is the n galaxies a thread will cover

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < n)
    {
        float rightAscension1 = galaxy1[index * PARAMETERS];
        float declination1 = galaxy1[index * PARAMETERS + 1];
        float radianToDegree = 180 / M_PI;

        for (int i = 0; i < n; i++)
        {
            float rightAscension2 = galaxy2[i * PARAMETERS];
            float declination2 = galaxy2[i * PARAMETERS + 1];

            // Calculate the angle between the two galaxies
            float angleRadians = angleBetweenGalaxies(rightAscension1, declination1, rightAscension2, declination2);
            float angleDegrees = angleRadians * radianToDegree;

            // Calculate the bin
            int bin = floor(angleDegrees * BINS_PER_DEGREE);

            // Increment the bin
            atomicAdd(&histogramBins[bin], 1);
        }
    }
}

int main(int argc, char *argv[])
{
    // Functions
    float arcminToRadian(float arcmin);
    float *readGalaxies(ifstream & file, int n);
    void printHistogram(unsigned int *histogramBins, unsigned long long &sum, int n, string path);

    int getDevice(int deviceno);
    void checkError(hipError_t error, string message);

    // Variables
    float *realGalaxies, *randomGalaxies;                  // CPU
    unsigned int *histogramDR, *histogramDD, *histogramRR; // CPU
    float *omega_values;                                   // CPU

    float *devRealGalaxies, *devRandomGalaxies;                     // GPU
    unsigned int *devHistogramDR, *devHistogramDD, *devHistogramRR; // GPU

    unsigned long long histogramDRsum, histogramDDsum, histogramRRsum;
    int nreal, nrandom, n;
    int numBlocks;

    double start, end, kerneltime;
    struct timeval _ttime;
    struct timezone _tzone;
    hipError_t myError;

    // Check arguments
    if (argc != 4)
    {
        cout << "Usage: a.out realData randomData output_data" << endl;
        return (-1);
    }

    // Select device
    if (getDevice(0) != 0)
    {
        cout << "Error in selecting device" << endl;
        return (-1);
    }

    kerneltime = 0.0;
    gettimeofday(&_ttime, &_tzone);
    start = (double)_ttime.tv_sec + (double)_ttime.tv_usec / 1000000.;

    // Read data
    ifstream realData(argv[1]);
    if (!realData.is_open())
    {
        cout << "Error opening real data file" << endl;
        return (-1);
    }

    ifstream randomData(argv[2]);
    if (!randomData.is_open())
    {
        cout << "Error opening random data file" << endl;
        return (-1);
    }

    ofstream output_data(argv[3]);
    if (!output_data.is_open())
    {
        cout << "Error opening output data file" << endl;
        return (-1);
    }

    realData >> nreal;
    randomData >> nrandom;

    // Check if the number of real and random galaxies is the same
    if (nreal != nrandom)
    {
        cout << "Error: Number of real and random galaxies must be the same" << endl;
        return (-1);
    }

    n = nreal;

    realGalaxies = readGalaxies(realData, n);
    randomGalaxies = readGalaxies(randomData, n);

    // Close files
    realData.close();
    randomData.close();

    // Initialization
    histogramDR = new unsigned int[TOTAL_BINS];
    histogramDD = new unsigned int[TOTAL_BINS];
    histogramRR = new unsigned int[TOTAL_BINS];
    omega_values = new float[TOTAL_BINS];
    histogramDRsum = 0;
    histogramDDsum = 0;
    histogramRRsum = 0;

    // Allocate memory on the GPU
    myError = hipMalloc((void **)&devRealGalaxies, n * PARAMETERS * sizeof(float));
    checkError(myError, "Error in allocating memory for real galaxies on the GPU");

    myError = hipMalloc((void **)&devRandomGalaxies, n * PARAMETERS * sizeof(float));
    checkError(myError, "Error in allocating memory for random galaxies on the GPU");

    myError = hipMalloc((void **)&devHistogramDR, TOTAL_BINS * sizeof(unsigned int));
    checkError(myError, "Error in allocating memory for histogram DR on the GPU");
    hipMemset(devHistogramDR, 0, TOTAL_BINS * sizeof(unsigned int));

    myError = hipMalloc((void **)&devHistogramDD, TOTAL_BINS * sizeof(unsigned int));
    checkError(myError, "Error in allocating memory for histogram DD on the GPU");
    hipMemset(devHistogramDD, 0, TOTAL_BINS * sizeof(unsigned int));

    myError = hipMalloc((void **)&devHistogramRR, TOTAL_BINS * sizeof(unsigned int));
    checkError(myError, "Error in allocating memory for histogram RR on the GPU");
    hipMemset(devHistogramRR, 0, TOTAL_BINS * sizeof(unsigned int));

    // Copy data to the GPU
    myError = hipMemcpy(devRealGalaxies, realGalaxies, n * PARAMETERS * sizeof(float), hipMemcpyHostToDevice);
    checkError(myError, "Error in copying real galaxies to the GPU");

    myError = hipMemcpy(devRandomGalaxies, randomGalaxies, n * PARAMETERS * sizeof(float), hipMemcpyHostToDevice);
    checkError(myError, "Error in copying random galaxies to the GPU");

    // Run the kernels on the GPU
    numBlocks = (n + NUM_THREADS - 1) / NUM_THREADS;
    calculateHistogram<<<numBlocks, NUM_THREADS>>>(devRealGalaxies, devRandomGalaxies, n, devHistogramDR);
    calculateHistogram<<<numBlocks, NUM_THREADS>>>(devRealGalaxies, devRealGalaxies, n, devHistogramDD);
    calculateHistogram<<<numBlocks, NUM_THREADS>>>(devRandomGalaxies, devRandomGalaxies, n, devHistogramRR);

    // Copy the results back to the CPU
    myError = hipMemcpy(histogramDR, devHistogramDR, TOTAL_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);
    checkError(myError, "Error in copying histogram DR to the CPU");

    myError = hipMemcpy(histogramDD, devHistogramDD, TOTAL_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);
    checkError(myError, "Error in copying histogram DD to the CPU");

    myError = hipMemcpy(histogramRR, devHistogramRR, TOTAL_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);
    checkError(myError, "Error in copying histogram RR to the CPU");

    // Print the histograms
    cout << "Histogram DR" << endl;
    printHistogram(histogramDR, histogramDRsum, n, "dr.txt");

    cout << "Histogram DD" << endl;
    printHistogram(histogramDD, histogramDDsum, n, "dd.txt");

    cout << "Histogram RR" << endl;
    printHistogram(histogramRR, histogramRRsum, n, "rr.txt");

    // Calculate omega values on the CPU
    for (int i = 0; i < TOTAL_BINS; i++)
    {
        float ddValue = histogramDD[i];
        float drValue = histogramDR[i];
        float rrValue = histogramRR[i];

        omega_values[i] = (ddValue - 2 * drValue + rrValue) / rrValue; // possible division by zero - expect nan value
    }

    // Write the omega values to the output file
    for (int i = 0; i < TOTAL_BINS; i++)
    {
        output_data << omega_values[i] << endl;
    }
    output_data.close();

    gettimeofday(&_ttime, &_tzone);
    end = (double)_ttime.tv_sec + (double)_ttime.tv_usec / 1000000.;
    kerneltime += end - start;
    cout << "Kernel time = " << kerneltime << " secs" << endl;

    // Delete memory
    delete[] realGalaxies;
    delete[] randomGalaxies;
    delete[] histogramDR;
    delete[] histogramDD;
    delete[] histogramRR;
    delete[] omega_values;

    hipFree(devRealGalaxies);
    hipFree(devRandomGalaxies);
    hipFree(devHistogramDR);
    hipFree(devHistogramDD);
    hipFree(devHistogramRR);

    return (0);
}

void printHistogram(unsigned int *histogramBins, unsigned long long &sum, int n, string path)
{
    for (int i = 0; i < TOTAL_BINS; i++)
    {
        sum += histogramBins[i];
    }

    cout << "Histogram sum: " << sum;

    if (sum == pow(n, 2))
        cout << " - Correct!" << endl;
    else
        cout << " - Incorrect!" << endl;

    ofstream file(path);
    if (!file.is_open())
    {
        cout << "Error opening file" << endl;
        return;
    }

    for (int i = 0; i < TOTAL_BINS; i++)
    {
        file << histogramBins[i] << endl;
    }
}

void checkError(hipError_t error, string message)
{
    if (error != hipSuccess)
    {
        cout << message << endl;
        cout << "Error: " << hipGetErrorString(error) << endl;
        exit(-1);
    }
}

float arcminToRadian(float arcmin)
{
    return arcmin * ARCMIN_TO_RADIAN;
}

float *readGalaxies(ifstream &file, int n)
{
    float *galaxies = new float[n * PARAMETERS]; // right ascension in arc minutes and declination in arc minutes - LINEARIZED
    for (int i = 0; i < n; i++)
    {
        float right_ascension, declination;
        file >> right_ascension >> declination;

        // Converting arc minutes to radians
        galaxies[i * PARAMETERS] = arcminToRadian(right_ascension);
        galaxies[i * PARAMETERS + 1] = arcminToRadian(declination);
    }
    return galaxies;
}

int getDevice(int deviceNo)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    cout << "Number of CUDA devices = " << deviceCount << endl;

    if (deviceCount < 0 || deviceCount > 128)
        return (-1);

    int device;

    hipSetDevice(deviceNo);
    hipGetDevice(&device);
    if (device != deviceNo)
        cout << "Unable to set device " << deviceNo << ", using device " << device << " instead" << endl;
    else
        cout << "Device " << device << " selected" << endl;

    return (0);
}
